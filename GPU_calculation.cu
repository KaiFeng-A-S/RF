#include "hip/hip_runtime.h"
#include "GPU_calculation.h"
#include <iostream>

__global__ void gini_count(int *_label_, int whole_size, int _num_, int *global_output){
    unsigned int t = threadIdx.x;
    unsigned int g = t + blockIdx.x * blockDim.x;
    unsigned int grid_size = blockDim.x * gridDim.x;
    int count[CLASS_NUMBER_MAX];
    __shared__ int partial_count[GINI_BLOCK_SIZE_MAX][CLASS_NUMBER_MAX];

    for(int i = 0; i < _num_; i++){count[i] = 0;}

    for(unsigned int i = g; i < whole_size; i += grid_size){
        count[_label_[i]]++;
    }

    //printf("%d, %d\n", t, count[0]);

    for(int i = 0; i < _num_; i++){
        partial_count[t][i] = count[i];
    }

    __syncthreads();

    //printf("p:%d, %d\n", t, partial_count[t][0]);

    for(int size = (blockDim.x / 2); size > 0; size >>= 1){
        if(t < size){
            //printf("if:%d, %d\n", t, size);
            for(int i = 0; i < _num_; i++){
                partial_count[t][i] += partial_count[t + size][i];
                //printf("pc:%d, %d, %d\n", t, partial_count[t][i], partial_count[t + size][i]);
            }
        }
        __syncthreads();
    }

    //printf("pp:%d, %d\n", t, partial_count[t][0]);

    if(t == 0){
        for(int i = 0; i < _num_; i++){
            global_output[(_num_ * blockIdx.x) + i] = partial_count[0][i];
        }
    }
}

__global__ void ginis_scan(int *_label_, float *_gini_, int whole_size, int _num_, int *count){
    unsigned int t = threadIdx.x;
    //unsigned int g = t + blockIdx.x * blockDim.x;
    //unsigned int grid_size = blockDim.x * gridDim.x;
    int ROUND = ((whole_size - 1) / blockDim.x) + 1;
    int last_count[CLASS_NUMBER_MAX];

    //printf("ROUND:%d\n", ROUND);

    __shared__ int XY[GINI_BLOCK_SIZE_MAX][CLASS_NUMBER_MAX];

    for(int i = 0; i < _num_; i++){
        XY[t][i] = 0;
        last_count[i] = 0;
    }

    for(int n = 0; n < ROUND; n++){

        //printf("n:%d\n", n);

        for(int i = 0; i < _num_; i++){
            XY[t][i] = 0;
        }

        int g = t + n * blockDim.x;
        if(g < whole_size){
            XY[t][_label_[g]] = 1;
        }

/*
        for(int i = 0; i < _num_; i++){
            printf("1:%d, %d\n", t, XY[t][i]);
        }
*/

        __syncthreads();

        for(int stride = 1; stride <= (blockDim.x / 2); stride <<= 1){
            int index = (t + 1) * stride * 2 - 1;
            if(index < blockDim.x){
                for(int i = 0; i < _num_; i++){
                    XY[index][i] += XY[index - stride][i];
                }
            }
            __syncthreads();
        }
        for(int stride  = (blockDim.x / 4); stride > 0; stride >>= 1){
            int index = (t + 1) * stride * 2 - 1;
            if(index + stride < blockDim.x){
                for(int i = 0; i < _num_; i++){
                    XY[index + stride][i] += XY[index][i];
                }
            }
            __syncthreads();
        }

        for(int i = 0; i < _num_; i++){
            XY[t][i] += last_count[i];
        }

/*
        for(int i = 0; i < _num_; i++){
            printf("2:%d, %d\n", t, XY[t][i]);
        }
*/

        float purity_l = 0.0;
        for(int i = 0; i < _num_; i++){
            float class_prop = (float) XY[t][i] / (g + 1);
            purity_l += class_prop * class_prop;
        }

        //printf("d:%d\n", t);

        float purity_r = 0.0;
        if(g < (whole_size - 1)){
            //printf("w_g:%d\n", whole_size - g);
            for(int i = 0; i < _num_; i++){
                //printf("count:%d, %d, %d\n", t, i, count[i]);
                float class_prop = (float) (count[i] - XY[t][i]) / (whole_size - g - 1);
                //printf("count:%d, %d, %d\n", t, i, count[i]);
                purity_r += class_prop * class_prop;
            }
        }

        //printf("dd:%d\n", t);
        
        if(g < whole_size){
            _gini_[g] = 1 - (((float) (g + 1) / whole_size) * purity_l + ((float) (whole_size - g - 1) / whole_size) * purity_r);
        }

/*
        for(int i = 0; i < _num_; i++){
            XY[t][i] = XY[blockDim.x - 1][i];
        }
*/

/*
        for(int i = 0; i < _num_; i++){
            printf("3:%d, %d\n", t, XY[t][i]);
        }

*/
        __syncthreads();

        //printf("n:%d\n", n);

        for(int i = 0; i < _num_; i++){
            last_count[i] = XY[blockDim.x - 1][i];
        }
    }
} 

void gini_calculation(int *_label_, int whole_size, int _num_, int BLOCKS, int THREADS, int *return_counts){
    int *dev_label;
    size_t size_labels = whole_size * sizeof(int);
    hipMalloc((void **)&dev_label, size_labels);
    hipMemcpy(dev_label, _label_, size_labels, hipMemcpyHostToDevice);

    int *dev_out;
    size_t size_out = _num_ * BLOCKS * sizeof(int);
    hipMalloc((void **)&dev_out, size_out);

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    gini_count<<<blocks, threads>>>(dev_label, whole_size, _num_, dev_out);
    hipDeviceSynchronize();

    int *out = (int *)malloc(size_out);
    hipMemcpy(out, dev_out, size_out, hipMemcpyDeviceToHost);
    hipFree(dev_label);
    hipFree(dev_out);

/*
    for(int i = 0; i < _num_ * BLOCKS; i++){
        std::cout<<out[i]<<std::endl;
    }
*/

    for(int i = 0; i < BLOCKS; i++){
        for(int j = 0; j < _num_; j++){
            return_counts[j] += out[(_num_ * i) + j];
        }
    }
}

void ginis(int *_label_, int *count, int whole_size, int _num_, int THREADS, float *return_ginis){
    int *dev_label;
    int *dev_count;
    size_t size_labels = whole_size * sizeof(int);
    size_t size_counts = _num_ * sizeof(int);
    hipMalloc((void **)&dev_label, size_labels);
    hipMalloc((void **)&dev_count, size_counts);
    hipMemcpy(dev_label, _label_, size_labels, hipMemcpyHostToDevice);
    hipMemcpy(dev_count, count, size_counts, hipMemcpyHostToDevice);

    float *dev_gini;
    size_t size_ginis = whole_size * sizeof(float);
    hipMalloc((void **)&dev_gini, size_ginis);    

    dim3 blocks(1, 1);
    dim3 threads(THREADS, 1);
    ginis_scan<<<blocks, threads>>>(dev_label, dev_gini, whole_size, _num_, dev_count);
    
    hipMemcpy(return_ginis, dev_gini, size_ginis, hipMemcpyDeviceToHost);
    hipFree(dev_label);
    hipFree(dev_count);
}
