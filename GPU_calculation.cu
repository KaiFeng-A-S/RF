#include "hip/hip_runtime.h"
#include "GPU_calculation.h"
#include <iostream>

__global__ void gini_count(int *_label_, int whole_size, int _num_, int *global_output){
    unsigned int t = threadIdx.x;
    unsigned int g = t + blockIdx.x * blockDim.x;
    unsigned int grid_size = blockDim.x * gridDim.x;
    int count[CLASS_NUMBER_MAX];
    __shared__ int partial_count[GINI_BLOCK_SIZE_MAX][CLASS_NUMBER_MAX];

    for(int i = 0; i < _num_; i++){count[i] = 0;}

    for(unsigned int i = g; i < whole_size; i += grid_size){
        count[_label_[i]]++;
    }

    //printf("%d, %d\n", t, count[0]);

    for(int i = 0; i < _num_; i++){
        partial_count[t][i] = count[i];
    }

    __syncthreads();

    //printf("p:%d, %d\n", t, partial_count[t][0]);

    for(int size = (blockDim.x / 2); size > 0; size >>= 1){
        if(t < size){
            //printf("if:%d, %d\n", t, size);
            for(int i = 0; i < _num_; i++){
                partial_count[t][i] += partial_count[t + size][i];
                //printf("pc:%d, %d, %d\n", t, partial_count[t][i], partial_count[t + size][i]);
            }
        }
        __syncthreads();
    }

    //printf("pp:%d, %d\n", t, partial_count[t][0]);

    if(t == 0){
        for(int i = 0; i < _num_; i++){
            global_output[(_num_ * blockIdx.x) + i] = partial_count[0][i];
        }
    }
}

__global__ void ginis_scan(int *_label_, float *_gini_, int whole_size, int _num_, int *count){
    unsigned int t = threadIdx.x;
    //unsigned int g = t + blockIdx.x * blockDim.x;
    //unsigned int grid_size = blockDim.x * gridDim.x;
    int ROUND = (whole_size / blockDim.x) + 1;
    __shared__ int XY[GINI_BLOCK_SIZE_MAX][CLASS_NUMBER_MAX];

    for(int i = 0; i < _num_; i++){
        XY[t][i] = 0;
    }

    for(int n = 0; n < ROUND; n++){

/*
        for(int i = 0; i < _num_; i++){
            XY[t][i] = 0;
        }
*/

        int g = t + n * blockDim.x;
        if(g < whole_size){
            XY[t][_label_[g]] += 1;
        }

        __syncthreads();

        for(int stride = 1; stride < (blockDim.x / 2), stride <<= 1){
            int index = (t + 1) * stride * 2 - 1;
            if(index < blockDim.x){
                for(int i = 0; i < _num_; i++){
                    XY[index][i] += XY[index - stride][i];
                }
            }
            __syncthreads();
        }
        for(int stride  = (blockDim.x / 4); stride > 0; stride >>= 1){
            int index = (t + 1) * stride * 2 - 1;
            if(index + stride < blockDim.x){
                for(int i = 0; i < _num_; i++){
                    XY[index + stride][i] += XY[index][i]
                }
            }
            __syncthreads();
        }

        float purity_l = 0.0;
        for(int i = 0; i <_num_; i++){
            float class_prop = (float) XY[t][i] / g;
            purity_l += class_prop * class_prop;
        }
        float purity_r = 0.0;
        for(int i = 0; i <_num_; i++){
            float class_prop = (float) (count[i] - XY[t][i]) / (whole_size - g);
            purity_r += class_prop * class_prop;
        }
        _gini_[g] = ((g / whole_size) * purity_l + ((whole_szie - g) / whole_size) * purity_r;

        for(int i = 0; i < _num_; i++){
            XY[t][i] = XY[blockDim.x][i];
        }
        __syncthreads();
    }
} 

void gini_calculation(int *_label_, int whole_size, int _num_, int BLOCKS, int THREADS, int *return_counts){
    int *dev_label;
    size_t size_labels = whole_size * sizeof(int);
    hipMalloc((void **)&dev_label, size_labels);
    hipMemcpy(dev_label, _label_, size_labels, hipMemcpyHostToDevice);

    int *dev_out;
    size_t size_out = _num_ * BLOCKS * sizeof(int);
    hipMalloc((void **)&dev_out, size_out);

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    gini_count<<<blocks, threads>>>(dev_label, whole_size, _num_, dev_out);
    hipDeviceSynchronize();

    int *out = (int *)malloc(size_out);
    hipMemcpy(out, dev_out, size_out, hipMemcpyDeviceToHost);
    hipFree(dev_label);
    hipFree(dev_out);

/*
    for(int i = 0; i < _num_ * BLOCKS; i++){
        std::cout<<out[i]<<std::endl;
    }
*/

    for(int i = 0; i < BLOCKS; i++){
        for(int j = 0; j < _num_; j++){
            return_counts[j] += out[(_num_ * i) + j];
        }
    }
}

void ginis(int *_label_, int *count, int whole_size, int _num_, int THREADS, float *return_ginis){
    int *dev_label;
    int *dev_count
    size_t size_labels = whole_size * sizeof(int);
    size_t size_counts = _num_ * sizeof(int);
    hipMalloc((void **)&dev_label, size_labels);
    hipMalloc((void **)&dev_count, size_count);
    hipMemcpy(dev_label, _label_, size_labels, hipMemcpyHostToDevice);
    hipMemcpy(dev_count, count, size_counts, hipMemcpyHostToDevice);

    float *dev_gini;
    size_t size_ginis = whole_size * sizeof(float);
    hipMalloc((void **)&dev_gini, size_ginis);    

    dim3 blocks(1, 1)
    dim3 threads(THREADS, 1);
    gini_scan<<<blocks, threads>>>(dev_label, count, dev_gini, whole_size, _num_);
    
    hipMemcpy(return_ginis, dev_gini, size_ginis, hipMemcpyDeviceToHost);
    hipFree(dev_label);
    hipFree(dev_count);
}
