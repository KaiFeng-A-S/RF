#include <hip/hip_runtime.h>
#include <float.h>

__global__ void bitonic_sort_step(float *dev_values, int j, int k){
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int ixj = i ^ j;
    
    if(ixj > i){
        if((i & k) == 0){
            if(dev_values[i] > dev_values[ixj]){
                float tmp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = tmp;
            }
        }
        else{
            if(dev_values[i] < dev_values[ixj]){
                float tmp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = tmp;
            }
        }
    }
}

__global__ void bitonic_sort_step_with_follower(float *dev_values, int *dev_followers, int j, int k){
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int ixj = i ^ j;

    if(ixj > i){
        if((i & k) == 0){
            if(dev_values[i] > dev_values[ixj]){
                float tmp_value = dev_values[i];
                int tmp_follower = dev_followers[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = tmp_value;
                dev_followers[i] = dev_followers[ixj];
                dev_followers[ixj] = tmp_follower;
            }
        }
        else{
            if(dev_values[i] < dev_values[ixj]){
                float tmp_value = dev_values[i];
                int tmp_follower = dev_followers[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = tmp_value;
                dev_followers[i] = dev_followers[ixj];
                dev_followers[ixj] = tmp_follower;
            }
        }
    }
}

__global__ void max_padding(float *dev_values, int length){
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

    if(i >= length){
        dev_values[i] = FLT_MAX;
    }
}

void bitonic_sort(float *values, int NUM_VALS, int BLOCKS, int THREADS){
    float *dev_values;
    size_t size = NUM_VALS * sizeof(float);

    hipMalloc((void **)&dev_values, size);
    hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    for(int k = 2; k <= NUM_VALS; k <<= 1){
        for(int j = (k >> 1); j > 0; j >>= 1){
            bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k);
        }
    }

    hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
    hipFree(dev_values);
}

void bitonic_sort_with_follower(float *values, int *followers, int NUM_ORIGINAL, int NUM_VALS, int BLOCKS, int THREADS){
    float *dev_values;
    int *dev_followers;
    size_t size_values = NUM_VALS * sizeof(float);
    size_t size_followers = NUM_VALS * sizeof(int);

    hipMalloc((void **)&dev_values, size_values);
    hipMemcpy(dev_values, values, size_followers, hipMemcpyHostToDevice);

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    max_padding<<<blocks, threads>>>(dev_values, NUM_ORIGINAL);

    for(int k = 2; k <= NUM_VALS; k <<= 1){
        for(int j = (k >> 1); j > 0; j >>= 1){
            bitonic_sort_step_with_follower<<<blocks, threads>>>(dev_values, dev_followers, j, k);
        }
    }

    hipMemcpy(values, dev_values, size_values, hipMemcpyDeviceToHost);
    hipMemcpy(followers, dev_followers, size_followers, hipMemcpyDeviceToHost);
    hipFree(dev_values);
    hipFree(dev_followers);
}
