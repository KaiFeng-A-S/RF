#include <hip/hip_runtime.h>
#include <float.h>
#include <limits.h>
#include <iostream>

__global__ void bitonic_sort_step(float *dev_values, int j, int k){
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int ixj = i ^ j;
    
    if(ixj > i){
        if((i & k) == 0){
            if(dev_values[i] > dev_values[ixj]){
                float tmp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = tmp;
            }
        }
        else{
            if(dev_values[i] < dev_values[ixj]){
                float tmp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = tmp;
            }
        }
    }
}

__global__ void bitonic_sort_step_with_follower(float *dev_values, int *dev_followers, int j, int k){
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int ixj = i ^ j;

    if(ixj > i){
        if((i & k) == 0){
            if(dev_values[i] > dev_values[ixj]){
                float tmp_value = dev_values[i];
                int tmp_follower = dev_followers[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = tmp_value;
                dev_followers[i] = dev_followers[ixj];
                dev_followers[ixj] = tmp_follower;
            }
        }
        else{
            if(dev_values[i] < dev_values[ixj]){
                float tmp_value = dev_values[i];
                int tmp_follower = dev_followers[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = tmp_value;
                dev_followers[i] = dev_followers[ixj];
                dev_followers[ixj] = tmp_follower;
            }
        }
    }
}

__global__ void max_padding(float *dev_values, int *dev_followers, int length){
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

    if(i >= length){
        dev_values[i] = FLT_MAX;
        dev_followers[i] = INT_MAX;
    }
}

void bitonic_sort(float *values, int NUM_VALS, int BLOCKS, int THREADS){
    float *dev_values;
    size_t size = NUM_VALS * sizeof(float);

    hipMalloc((void **)&dev_values, size);
    hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    for(int k = 2; k <= NUM_VALS; k <<= 1){
        for(int j = (k >> 1); j > 0; j >>= 1){
            bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k);
        }
    }

    hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
    hipFree(dev_values);
}

void bitonic_sort_with_follower(float *values, int *followers, int NUM_ORIGINAL, int NUM_VALS, int BLOCKS, int THREADS){
    float *dev_values;
    int *dev_followers;
    size_t size_values = NUM_VALS * sizeof(float);
    size_t size_followers = NUM_VALS * sizeof(int);

    hipError_t error = hipMalloc((void **)&dev_values, size_values);
    //std::cout<<error<<std::endl;
    error = hipMalloc((void **)&dev_followers, size_followers);
    //std::cout<<error<<std::endl;
    error = hipMemcpy(dev_values, values, size_values, hipMemcpyHostToDevice);
    //std::cout<<error<<std::endl;
    error = hipMemcpy(dev_followers, followers, size_followers, hipMemcpyHostToDevice);
    //std::cout<<error<<std::endl;

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    max_padding<<<blocks, threads>>>(dev_values, dev_followers, NUM_ORIGINAL);

    for(int k = 2; k <= NUM_VALS; k <<= 1){
        for(int j = (k >> 1); j > 0; j >>= 1){
            bitonic_sort_step_with_follower<<<blocks, threads>>>(dev_values, dev_followers, j, k);
        }
    }

    error = hipMemcpy(values, dev_values, size_values, hipMemcpyDeviceToHost);
    //std::cout<<error<<std::endl;
    error = hipMemcpy(followers, dev_followers, size_followers, hipMemcpyDeviceToHost);
    //std::cout<<dev_values<<std::endl;
    //std::cout<<dev_followers<<std::endl;
    error = hipFree(dev_values);
    std::cout<<error<<std::endl;
    hipFree(dev_followers);
    //std::cout<<error<<std::endl;
}
