#include <hip/hip_runtime.h>

__global__ void bitonic_sort_step(float *dev_values, int j, int k){
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int ixj = i^j;
    
    if(ixj > i){
        if((i & k) == 0){
            if(dev_values[i] > dev_values[ixj]){
                float tmp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = tmp;
            }
        }
        else{
            if(dev_values[i] < dev_values[ixj]){
                float tmp = dev_values[i];
                dev_values[i] = dev_values[ixj];
                dev_values[ixj] = tmp;
            }
        }
    }
}

void bitonic_sort(float *values, int NUM_VALS, int BLOCKS, int THREADS){
    float *dev_values;
    size_t size = NUM_VALS * sizeof(float);

    hipMalloc((void**) &dev_values, size);
    hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

    dim3 blocks(BLOCKS, 1);
    dim3 threads(THREADS, 1);

    for(int k = 2; k <= NUM_VALS; k <<= 1){
        for(int j = k >> 1; j > 0; j >>= 1){
            bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k);
        }
    }

    hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
    hipFree(dev_values);
}
